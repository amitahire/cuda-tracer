#include "hip/hip_runtime.h"
#include <iostream>
#include <float.h>
#include <fstream>
#include <algorithm>
#include <ctime>
#include <SDL.h>

// For the CUDA runtime routines (prefixed with "cuda_")
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime_api.h>

#include "sphere.h"
#include "hitable_list.h"
#include "renderer.h"

using namespace std;

struct window {
	SDL_Window* w_screen;
	SDL_Renderer* w_renderer;
	SDL_Texture* w_texture;
	unsigned int *w_pixels;
	int w_nx;
	int w_ny;

	bool quit;
	bool mouse_drag;

	float theta = 80 * M_PI / 180;
	float phi = 45 * M_PI / 180;
	const float delta = 1 * M_PI / 180;

	renderer& w_r;
	camera *w_cam;

	window(int nx, int ny, renderer &r, camera *cam): w_nx(nx), w_ny(ny), w_r(r), w_cam(cam) {
		SDL_Init(SDL_INIT_VIDEO);

		w_screen = SDL_CreateWindow("Voxel Tracer (rendering)", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, nx, ny, 0);
		w_renderer = SDL_CreateRenderer(w_screen, -1, 0);
		w_texture = SDL_CreateTexture(w_renderer, SDL_PIXELFORMAT_ARGB8888, SDL_TEXTUREACCESS_STATIC, nx, ny);
		w_pixels = new unsigned int[nx*ny];

		cam->look_from(theta, phi);
		w_r.update_camera();
	}

	void destroy() {
		SDL_DestroyTexture(w_texture);
		SDL_DestroyRenderer(w_renderer);
		SDL_DestroyWindow(w_screen);
		delete[] w_pixels;

		SDL_Quit();
	}

	void update_pixels() {
		for (int x = 0; x < w_nx; x++)
		{
			for (int y = 0; y < w_ny; y++)
			{
				vec3 col = w_r.get_pixel_color(x, y);
				col = vec3(sqrtf(col[0]), sqrtf(col[1]), sqrtf(col[2]));
				int ir = min(255, int(255.99*col.r()));
				int ig = min(255, int(255.99*col.g()));
				int ib = min(255, int(255.99*col.b()));
				w_pixels[(w_ny - 1 - y)*w_nx + x] = (ir << 16) + (ig << 8) + ib;
			}
		}
		SDL_UpdateTexture(w_texture, NULL, w_pixels, w_nx * sizeof(unsigned int));
		//SDL_RenderClear(w_renderer);
		SDL_RenderCopy(w_renderer, w_texture, NULL, NULL);
		SDL_RenderPresent(w_renderer);
	}

	void poll_events() {
		SDL_Event event;
		while (SDL_PollEvent(&event)) {
			switch (event.type) {
			case SDL_QUIT:
				quit = true;
				break;
			case SDL_MOUSEMOTION:
				if (mouse_drag) {
					int mx = event.motion.xrel;
					int my = event.motion.yrel;
					theta += -my*delta;
					if (theta < delta) theta = delta;
					if (theta >(M_PI_2 - delta)) theta = M_PI_2 - delta;
					phi += -mx*delta;
					w_cam->look_from(theta, phi);
					w_r.update_camera();
				}
				break;
			case SDL_MOUSEBUTTONDOWN:
				mouse_drag = true;
				break;
			case SDL_MOUSEBUTTONUP:
				mouse_drag = false;
				break;
			}
		}
	}

	void set_title(char *title) {
		SDL_SetWindowTitle(w_screen, title);
	}

	void wait_to_quit() {
		SDL_Event event;
		while (!quit) {
			SDL_WaitEvent(&event);
			quit = event.type == SDL_QUIT;
		}
	}
};

void random_scene(hitable_list **scene, camera **cam, float aspect)
{
    int n = 500;
    hitable **list = new hitable*[n+1];
    list[0] =  new sphere(vec3(0,-1000,0), 1000, make_lambertian(vec3(0.5, 0.5, 0.5)));
    int i = 1;
    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            float choose_mat = drand48();
            vec3 center(a+0.9*drand48(),0.2,b+0.9*drand48());
            if ((center-vec3(4,0.2,0)).length() > 0.9) {
                if (choose_mat < 0.8) {  // diffuse
                    list[i++] = new sphere(center, 0.2, make_lambertian(vec3(drand48()*drand48(), drand48()*drand48(), drand48()*drand48())));
                }
                else if (choose_mat < 0.95) { // metal
                    list[i++] = new sphere(center, 0.2,
                            make_metal(vec3(0.5*(1 + drand48()), 0.5*(1 + drand48()), 0.5*(1 + drand48())),  0.5*drand48()));
                }
                else {  // glass
                    list[i++] = new sphere(center, 0.2, make_dielectric(1.5));
                }
            }
        }
    }

    list[i++] = new sphere(vec3(0, 1, 0), 1.0, make_dielectric(1.5));
    list[i++] = new sphere(vec3(-4, 1, 0), 1.0, make_lambertian(vec3(0.4, 0.2, 0.1)));
    list[i++] = new sphere(vec3(4, 1, 0), 1.0, make_metal(vec3(0.7, 0.6, 0.5), 0.0));
	list[i++] = new sphere(vec3(10, 10, 10), 0.5, make_diffuse_light(vec3(200, 200, 100)));

	*scene = new hitable_list(list, i);
	*cam = new camera(vec3(13, 2, 3), vec3(0, 0, 0), vec3(0, 1, 0), 20, aspect, 0.1, 10.0);
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
	bool print_progress = false;
	bool write_image = false;
	bool show_window = true;

	const unsigned int scene_size = 500;
	const int nx = 600;
	const int ny = 300;
	const int ns = 10000;
	hitable_list *world;
	camera *cam;
	
	random_scene(&world, &cam, float(nx) / float(ny));
	
	renderer r(cam, world, nx, ny, ns, 50, 0.001);
	r.prepare_kernel();

	window *w;
	if (show_window) {
		w = new window(nx, ny, r, cam);
	}

    clock_t begin = clock();

	unsigned int iteration = 0;
	unsigned int total_rays = 0;
	while ((show_window && !w->quit) || (!show_window && r.numrays() > 0))
	{
		if (r.numrays() > 0)
		{
			total_rays += r.numrays();
			if (print_progress && iteration % 100 == 0)
			{
				cout << "iteration " << iteration << "(" << r.numrays() << " rays)\r";
				cout.flush();
			}

			// compute ray-world intersections
			r.run_kernel();
			// compact active rays
			r.compact_rays();
		}

		if (show_window) {
			w->update_pixels();
			w->poll_events();
		}

		++iteration;

	}

    clock_t end = clock();
	printf("rendering %d rays, duration %.2f seconds\nkernel %.2f seconds\ngenerate %.2f seconds\ncompact %.2f seconds\n",
		total_rays,
		double(end - begin) / CLOCKS_PER_SEC,
		double(r.kernel) / CLOCKS_PER_SEC,
		double(r.generate) / CLOCKS_PER_SEC,
		double(r.compact) / CLOCKS_PER_SEC);
	cout.flush();

	if (show_window) {
		w->set_title("Voxel Tracer");
		w->wait_to_quit();
		w->destroy();
	}
  
	if (write_image) {
		// generate final image
		ofstream image;
		image.open("picture.ppm");
		image << "P3\n" << nx << " " << ny << "\n255\n";
		unsigned int sample_idx = 0;
		for (int j = ny - 1; j >= 0; j--)
		{
			for (int i = 0; i < nx; ++i, sample_idx++)
			{
				vec3 col = r.get_pixel_color(i, j);
				col = vec3(sqrtf(col[0]), sqrtf(col[1]), sqrtf(col[2]));
				int ir = min(255, int(255.99*col.r()));
				int ig = min(255, int(255.99*col.g()));
				int ib = min(255, int(255.99*col.b()));

				image << ir << " " << ig << " " << ib << "\n";
			}
		}
	}
	
	r.destroy();

    return 0;
}

