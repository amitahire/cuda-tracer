#include "hip/hip_runtime.h"
#include "pdf.h"

float pdf::value(const float3& direction) const {
	float cosine;
	switch (type) {
	case COSINE:
		cosine = dot(normalize(direction), uvw.w());
		if (cosine > 0)
			return cosine / M_PI;
		return 0;
	case HITABLE:
		return ptr->pdf_value(o, direction);
	case MIXTURE:
		return 0.5*p[0]->value(direction) + 0.5*p[1]->value(direction);
	}

	return 0;
}

__device__ float3 pdf::generate(seed_t seed) const {
	switch (type) {
	case COSINE:
		return uvw.local(random_cosine_direction(seed));
	case HITABLE:
		return ptr->random(seed, o);
	case MIXTURE:
		return (cu_drand48(seed) < 0.5) ? p[0]->generate(seed) : p[1]->generate(seed);
	}

	return make_float3(1, 0, 0); // we should throw an error
}
